#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void kernel()
{
	// ...
	int kk = 1;
}

extern "C" void launch_kernel()
{
	printf("RUN CUDA KERNEL\n");
	kernel << <1, 1 >> >();
}